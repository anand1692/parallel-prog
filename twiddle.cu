#include<stdio.h>
#include<hip/hip_runtime.h>

/* Producing twiddle factors */
#define NUM_OF_X_THREADS 10
#define NUM_OF_Y_THREADS 10

__global__ void inputKernel(float *x, int N)
{
    int ix   = blockIdx.x * blockDim.x + threadIdx.x;
    int iy   = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = iy * NUM_OF_X_THREADS + ix;

    if (idx < N)
        x[idx]  = x[idx] + (float)idx;
}

__global__ void factorKernel(float *w, int N)
{
    int ix  = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = ix * 2;
    int izx = N + idx;

    const float pi = 3.1415;
    float aw = (2.0 * pi) / (float)N;
    float arg = aw * (float)ix;

    /* Twiddle factors are symmetric along N/2. with change in sign, due to 180 degree phase change */
    if (idx < N) {
        w[idx] = cos(arg);
        w[idx + 1] = sin(arg);
        w[izx] = (-1) * w[idx];
        w[izx+1] = (-1) * w[idx + 1];
    }
}

__global__ void twiddleRealKernel(float *wr, float *w, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = 0, index;

    if (idx < N) {
        if (idx == 0) {
            for (i = 0; i < N; i++)
                wr[idx * N + i] = 1;
            } else {
                wr[idx * N + 0] = 1;
                for (i = 1; i < N; i++) {
                    index = (idx * i) % N;
                    wr[idx * N + i] = w[index * 2];
                }
            }
    }
}

__global__ void twiddleImgKernel(float *wi, float *w, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i, index;

    if (idx < N) {
        if (idx == 0) {
          for (i = 0; i < N; i++)
          wi[idx * N + i] = 0;
        } else {
             wi[idx * N + 0] = 0;
             for (i = 1; i < N; i++) {
                index = (idx * i) % N;
                wi[idx * N + i] = w[index * 2 + 1];

             }
        }
    }
}

int main(int agrc, char** argv)
{
    float *x, *w, *w_r, *w_i;
    float *d_x, *d_w, *dw_r, *dw_i;

    int N = 10000, n = N/2;

    x = (float *)malloc(N * sizeof(float));
    w = (float *)malloc(2 * N * sizeof(float));
    w_r = (float *)malloc(N * N * sizeof(float));
    w_i = (float *)malloc(N * N * sizeof(float));
    dim3 numberOfThreads(NUM_OF_X_THREADS, NUM_OF_Y_THREADS);
    dim3 numberOfBlocks( (100 + NUM_OF_X_THREADS -1)/NUM_OF_X_THREADS,
                         (100 + NUM_OF_Y_THREADS - 1)/NUM_OF_Y_THREADS );

    hipMalloc((void **)&d_x, N * sizeof(float));
    hipMalloc((void **)&d_w, 2 * N * sizeof(float));
    hipMalloc((void **)&dw_r, N * N * sizeof(float));
    hipMalloc((void **)&dw_i, N * N * sizeof(float));

    hipMemset(d_x, 0, N * sizeof(float));
    hipMemset(d_w, 0, 2 * N * sizeof(float));
    hipMemset(dw_r, 0, N * N * sizeof(float));
    hipMemset(dw_i, 0, N * N * sizeof(float));

    inputKernel<<<numberOfBlocks, numberOfThreads>>>(d_x, N);
    hipMemcpy(x, d_x, N * sizeof(float), hipMemcpyDeviceToHost);
    printf("%f\n",x[100]);
    // Calculating factor
    factorKernel<<<n/512, 512>>>(d_w, (float)N);
    hipMemcpy(w, d_w, 2 * N * sizeof(float), hipMemcpyDeviceToHost);
    printf("%f %f\n", w[5], w[10005]);

    // Calculating twiddle real matrix
    twiddleRealKernel<<<n/512, 512>>>(dw_r, d_w, N);
    hipMemcpy(w_r, dw_r, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Calculating twiddle imaginary matrix
    twiddleImgKernel<<<n/512, 512>>>(dw_i, d_w, N);
    hipMemcpy(w_i, dw_i, N * N * sizeof(float), hipMemcpyDeviceToHost);
    /*  int i,j;
        for(i = 0; i < 50; i++)
        {
            for(j = 0; j < 50; j++) {
                printf("%f \t", w_r[i*N + j]);
            }
            printf("\n");
        }
      printf("*********************************************************************************\n");
      for(i = 0; i < 50; i++) {
        for(j = 0; j < 50; j++) {
          printf("%f \t", w_i[i*N + j]);
        }
        printf("\n");
      }
*/
  return 0;
}




